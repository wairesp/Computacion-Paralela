#include "hip/hip_runtime.h"
%%cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define DIM 8
#define BlockSize 4

__global__ void multi(int *B, int *C, int *A, int width)
{
    int cvalue = 0;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;


    if (row > width || col > width) return;

    for (int e = 0; e < width; ++e){
        cvalue += B[row * width + e] * C[e * width + col];
    }
    A[row * width + col] = cvalue;

}

void matrixmulti(int B[][DIM], int C[][DIM], int A[][DIM]){
    int *dev_a, *dev_b, *dev_c;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate memory on global memory of gpu
    hipMalloc((void**)&dev_a, ((DIM)*(DIM))*sizeof(int));
    hipMalloc((void**)&dev_b, ((DIM)*(DIM))*sizeof(int));
    hipMalloc((void**)&dev_c, ((DIM)*(DIM))*sizeof(int));

    //Copy array B and C on device allocated memory
    hipMemcpy(dev_a, B, ((DIM * DIM)) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, C, ((DIM * DIM)) * sizeof(int), hipMemcpyHostToDevice);

    //two dimension threads
    dim3 dimBlock(BlockSize, BlockSize);
    dim3 dimGrid((DIM + dimBlock.x - 1) / dimBlock.x, (DIM + dimBlock.y - 1) / dimBlock.y);

    //call the kernel function multi
    hipEventRecord(start);
    multi << < dimGrid, dimBlock >> >(dev_a, dev_b, dev_c, DIM);
    hipEventRecord(stop);

    //retrieve array A from device memory
    hipMemcpy(A, dev_c, ((DIM * DIM)) * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    /*for (int i = 0; i < DIM; i++){
        for (int j = 0; j < DIM; j++){
            printf("A(%d,%d) = %d \n", i, j, A[i][j]);
        }
    }*/

    //free the memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}


int main(){

	srand(time(0));
	auto A = new int[DIM][DIM];
	auto B = new int[DIM][DIM];
	auto C = new int[DIM][DIM];

	//populate the arrays A and B
	for (int i = 0; i<DIM; i++){
		for (int j = 0; j < DIM; j++){
			A[i][j] = rand() % 10;
			B[i][j] = rand() % 10;
		}
	}

  printf("VALORES DE MATRIZ B \n");
	for (int y = 0; y < DIM; y++)
	{
		for (int x = 0; x < DIM; x++)
		{
			printf("%d", B[y][x]);
      printf(" ");
		}
		printf(" \n");
	}
  
  printf("UCSP\nVALORES DE MATRIZ C \n");
	for (int y = 0; y < DIM; y++)
	{
		for (int x = 0; x < DIM; x++)
		{
			printf("%d", C[y][x]);
      printf(" ");
		}
		printf(" \n");
	}

	
  matrixmulti(B,C,A);

  printf("VALORES DE MATRIZ A \n");
	for (int y = 0; y < DIM; y++)
	{
		for (int x = 0; x < DIM; x++)
		{
			printf("%d", A[y][x]);
      printf(" ");
		}
		printf(" \n");
	}

	//delete arrays
	delete[]A;
	delete[]B;
	delete[]C;
}